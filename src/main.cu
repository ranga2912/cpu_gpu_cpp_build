#include <iostream>
#include <hip/hip_runtime.h>

// GPU kernel to perform addition
__global__ void addGPU(int* a, int* b, int* result) {
    *result = *a + *b;
}

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << hipGetErrorString(err) << std::endl; \
            exit(1); \
        } \
    } while(0)

// Function to perform addition on GPU
int addOnGPU(int a, int b) {
    int *d_a, *d_b, *d_result;
    int result = 0;  // Initialize to avoid undefined behavior

    CUDA_CHECK(hipMalloc(&d_a, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_b, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_result, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice));

    addGPU<<<1, 1>>>(d_a, d_b, d_result);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_result));

    return result;
}

int main() {
    int variable1 = 5000 ;
    int variable2 = 10000;
    int sum;

    // Use GPU calculation
    sum = addOnGPU(variable1, variable2);
    std::cout << "=== GPU Addition Demo ===" << std::endl;
    std::cout << "GPU calculation: " << variable1 << " + " << variable2 << " = " << sum << std::endl;
    std::cout << "GPU acceleration working!" << std::endl;

    return 0;
}
