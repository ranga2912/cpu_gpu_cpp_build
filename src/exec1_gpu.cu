#include <iostream>
#include <hip/hip_runtime.h>

// GPU kernel to perform addition
__global__ void addGPU(int* a, int* b, int* result) {
    int idx = threadIdx.x;
    if (idx == 0) {  // Only one thread needs to do this simple operation
        *result = *a + *b;
    }
}

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << hipGetErrorString(err) << std::endl; \
            exit(1); \
        } \
    } while(0)

int main() {
    // Declare and initialize the variables
    int variable1 = 5;
    int variable2 = 10;
    
    // GPU calculation ONLY
    int *d_a, *d_b, *d_result;
    int result;
    
    // Allocate GPU memory
    CUDA_CHECK(hipMalloc(&d_a, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_b, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_result, sizeof(int)));
    
    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_a, &variable1, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, &variable2, sizeof(int), hipMemcpyHostToDevice));
    
    // Launch GPU kernel
    addGPU<<<1, 1>>>(d_a, d_b, d_result);
    CUDA_CHECK(hipGetLastError());
    
    // Wait for GPU to finish
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy result back from GPU
    CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost));
    
    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    
    // Print the GPU result
    std::cout << "=== GPU Addition Demo ===" << std::endl;
    std::cout << "GPU calculation: " << variable1 << " + " << variable2 << " = " << result << std::endl;
    std::cout << "GPU acceleration working!" << std::endl;
    
    return 0;
}
